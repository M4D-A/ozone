#include "hip/hip_runtime.h"
#include "frequency.h"

__global__ auto iota(std::span<uint8_t> data) -> void {
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < data.size()) {
    data[tid] = tid;
  }
}
__global__ auto byte_popc(std::span<const uint8_t> data, std::span<uint8_t> counts) -> void {
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < data.size()) {
    counts[tid] = __popc(data[tid]);
  }
}
