#include "cuda_tests/frequency/frequency.h"
#include <cstdint>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ranges>
#include <span>

namespace rv = std::ranges::views;

auto main() -> int {
  int N = 25600;

  uint8_t *data_ptr = nullptr;
  uint8_t *output_ptr = nullptr;
  hipMallocManaged(&data_ptr, N * sizeof(uint8_t));
  hipMallocManaged(&output_ptr, N * sizeof(uint8_t));

  std::span<uint8_t> data(data_ptr, N);
  std::span<uint8_t> output(data_ptr, N);

  int block_size = 256;
  int blocks_num = (N - 1) / block_size + 1;

  iota<<<blocks_num, block_size>>>(data);
  byte_popc<<<blocks_num, block_size>>>(data, output);
  hipDeviceSynchronize();

  for (auto count : output | rv::take(10)) {
    std::cout << (int)count << ", ";
  }
  std::cout << "\n";

  for (auto count : output | rv::reverse | rv::take(10)) {
    std::cout << (int)count << ", ";
  }
  std::cout << "\n";

  return 0;
}
